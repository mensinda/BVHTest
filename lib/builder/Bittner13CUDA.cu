#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2018 Daniel Mensinger
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "base/BVH.hpp"
#include "base/BVHPatch.hpp"
#include "cuda/cudaFN.hpp"
#include "Bittner13CUDA.hpp"
#include "CUDAHeap.hpp"
#include <cmath>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>
#include <iostream>

using namespace glm;
using namespace std;
using namespace BVHTest;
using namespace BVHTest::base;
using namespace BVHTest::cuda;

#define CUDA_RUN(call)                                                                                                 \
  lRes = call;                                                                                                         \
  if (lRes != hipSuccess) {                                                                                           \
    cout << "CUDA ERROR (" << __FILE__ << ":" << __LINE__ << "): " << hipGetErrorString(lRes) << endl;                \
    goto error;                                                                                                        \
  }

#define ALLOCATE(ptr, num, type) CUDA_RUN(hipMalloc(ptr, num * sizeof(type)));
#define FREE(ptr, num)                                                                                                 \
  hipFree(ptr);                                                                                                       \
  ptr = nullptr;                                                                                                       \
  num = 0;

#define FREE2(ptr)                                                                                                     \
  hipFree(ptr);                                                                                                       \
  ptr = nullptr;

#define IF_LOCK(N, VAL) if (atomicCAS(_flags + N, 0u, VAL) == 0u)
#define IF_NOT_LOCK(N, VAL) if (atomicCAS(_flags + N, 0u, VAL) != 0u)
#define RELEASE_LOCK(N) atomicExch(_flags + N, 0u);
#define RELEASE_LOCK_S(N, VAL) atomicCAS(_flags + N, VAL, 0u);

struct CUBLeafSelect {
  BVHNode *nodes;

  HIPCUB_RUNTIME_FUNCTION __forceinline__ CUBLeafSelect(BVHNode *_n) : nodes(_n) {}

  __device__ __forceinline__ bool operator()(const uint32_t &a) const { return nodes[a].numChildren == 0; }
};

__global__ void kResetTodoData(uint32_t *_nodes, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;
  for (uint32_t i = index; i < _num; i += stride) { _nodes[i] = i; }
}

__global__ void kInitPatches(PATCH *_patches, BVH *_bvh, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;
  for (uint32_t i = index; i < _num; i += stride) { new (_patches + i) PATCH(_bvh); }
}


/*  ______                                                _            _                     _     */
/*  | ___ \                                              | |          (_)                   | |    */
/*  | |_/ /___ _ __ ___   _____   _____    __ _ _ __   __| |  _ __ ___ _ _ __  ___  ___ _ __| |_   */
/*  |    // _ \ '_ ` _ \ / _ \ \ / / _ \  / _` | '_ \ / _` | | '__/ _ \ | '_ \/ __|/ _ \ '__| __|  */
/*  | |\ \  __/ | | | | | (_) \ V /  __/ | (_| | | | | (_| | | | |  __/ | | | \__ \  __/ |  | |_   */
/*  \_| \_\___|_| |_| |_|\___/ \_/ \___|  \__,_|_| |_|\__,_| |_|  \___|_|_| |_|___/\___|_|   \__|  */
/*                                                                                                 */



struct CUDAHelperStruct {
  uint32_t node;
  float    cost;
  uint32_t level;

  __device__ __forceinline__ bool operator<(CUDAHelperStruct const &_b) const noexcept { return cost > _b.cost; }
};

struct CUDANodeLevel {
  uint32_t node;
  uint32_t level;
};

struct CUDA_RM_RES {
  struct NodePair {
    uint32_t n1;
    uint32_t n2;
  };

  bool     res;
  NodePair toInsert;
  NodePair unused;
  NodePair grandParentAndSibling;
};

struct CUDA_INS_RES {
  bool     res;
  uint32_t best;
  uint32_t root;
};

__device__ CUDANodeLevel findNodeForReinsertion(uint32_t _n, PATCH &_bvh) {
  float             lBestCost      = HUGE_VALF;
  CUDANodeLevel     lBestNodeIndex = {0, 0};
  BVHNode const *   lNode          = _bvh[_n];
  AABB const &      lNodeBBox      = lNode->bbox;
  float             lSArea         = lNode->surfaceArea;
  uint32_t          lSize          = 1;
  CUDAHelperStruct  lPQ[CUDA_QUEUE_SIZE];
  CUDAHelperStruct *lBegin = lPQ;

  lPQ[0] = {_bvh.root(), 0.0f, 0};
  while (lSize > 0) {
    CUDAHelperStruct lCurr     = lPQ[0];
    BVHNode *        lCurrNode = _bvh[lCurr.node];
    auto             lBBox     = _bvh.getAABB(lCurr.node, lCurr.level);
    CUDA_pop_heap(lBegin, lBegin + lSize);
    lSize--;

    if ((lCurr.cost + lSArea) >= lBestCost) {
      // Early termination - not possible to further optimize
      break;
    }

    lBBox.box.mergeWith(lNodeBBox);
    float lDirectCost = lBBox.box.surfaceArea();
    float lTotalCost  = lCurr.cost + lDirectCost;
    if (lTotalCost < lBestCost) {
      // Merging here improves the total SAH cost
      lBestCost      = lTotalCost;
      lBestNodeIndex = {lCurr.node, lCurr.level};
    }

    float lNewInduced = lTotalCost - lBBox.sarea;
    if ((lNewInduced + lSArea) < lBestCost) {
      if (!lCurrNode->isLeaf()) {
        assert(lSize + 2 < CUDA_QUEUE_SIZE);
        lPQ[lSize + 0] = {lCurrNode->left, lNewInduced, lCurr.level + 1};
        lPQ[lSize + 1] = {lCurrNode->right, lNewInduced, lCurr.level + 1};
        CUDA_push_heap(lBegin, lBegin + lSize + 1);
        CUDA_push_heap(lBegin, lBegin + lSize + 2);
        lSize += 2;
      }
    }
  }

  return lBestNodeIndex;
}


__device__ CUDA_RM_RES removeNode(uint32_t _node, PATCH &_bvh, uint32_t *_flags, uint32_t _lockID) {
  CUDA_RM_RES lFalse = {false, {0, 0}, {0, 0}, {0, 0}};
  if (_bvh[_node]->isLeaf() || _node == _bvh.root()) { return lFalse; }
  assert(_node != 0);

  IF_NOT_LOCK(_node, _lockID) { return lFalse; }

  BVHNode *lNode         = _bvh.patchNode(_node);
  uint32_t lSiblingIndex = _bvh.sibling(*lNode);
  uint32_t lParentIndex  = lNode->parent;

  if (lParentIndex == _bvh.root()) {
    RELEASE_LOCK_S(_node, _lockID);
    return lFalse;
  } // Can not remove node with this algorithm

  assert(lParentIndex != 0);


  IF_NOT_LOCK(lSiblingIndex, _lockID) {
    RELEASE_LOCK_S(_node, _lockID);
    return lFalse;
  }
  BVHNode *lSibling = _bvh.patchNode(lSiblingIndex);

  IF_NOT_LOCK(lParentIndex, _lockID) {
    RELEASE_LOCK_S(_node, _lockID);
    RELEASE_LOCK_S(lSiblingIndex, _lockID);
    return lFalse;
  }
  BVHNode *lParent           = _bvh.patchNode(lParentIndex);
  uint32_t lGrandParentIndex = lParent->parent;

  IF_NOT_LOCK(lGrandParentIndex, _lockID) {
    RELEASE_LOCK_S(_node, _lockID);
    RELEASE_LOCK_S(lSiblingIndex, _lockID);
    RELEASE_LOCK_S(lParentIndex, _lockID);
    return lFalse;
  }
  BVHNode *lGrandParent = _bvh.patchNode(lGrandParentIndex);

  IF_NOT_LOCK(lNode->left, _lockID) {
    RELEASE_LOCK_S(_node, _lockID);
    RELEASE_LOCK_S(lSiblingIndex, _lockID);
    RELEASE_LOCK_S(lParentIndex, _lockID);
    RELEASE_LOCK_S(lGrandParentIndex, _lockID);
    return lFalse;
  }

  IF_NOT_LOCK(lNode->right, _lockID) {
    RELEASE_LOCK_S(_node, _lockID);
    RELEASE_LOCK_S(lSiblingIndex, _lockID);
    RELEASE_LOCK_S(lParentIndex, _lockID);
    RELEASE_LOCK_S(lGrandParentIndex, _lockID);
    RELEASE_LOCK_S(lNode->left, _lockID);
    return lFalse;
  }

  BVHNode *lLeft  = _bvh.patchNode(lNode->left);
  BVHNode *lRight = _bvh.patchNode(lNode->right);

  // FREE LIST:   lNode, lParent
  // INSERT LIST: lLeft, lRight

  float lLeftSA  = lLeft->surfaceArea;
  float lRightSA = lRight->surfaceArea;

  // Remove nodes
  if (lParent->isLeftChild()) {
    lGrandParent->left = lSiblingIndex;
    lSibling->isLeft   = TRUE;
    lSibling->parent   = lGrandParentIndex;
  } else {
    lGrandParent->right = lSiblingIndex;
    lSibling->isLeft    = FALSE;
    lSibling->parent    = lGrandParentIndex;
  }

  // update Bounding Boxes (temporary)
  _bvh.patchAABBFrom(lGrandParentIndex);

  if (lLeftSA > lRightSA) {
    return {true, {lNode->left, lNode->right}, {_node, lParentIndex}, {lGrandParentIndex, lSiblingIndex}};
  } else {
    return {true, {lNode->right, lNode->left}, {_node, lParentIndex}, {lGrandParentIndex, lSiblingIndex}};
  }
}


__device__ CUDA_INS_RES
           reinsert(uint32_t _node, uint32_t _unused, PATCH &_bvh, bool _update, uint32_t *_flags, uint32_t _lockID) {
  CUDANodeLevel lRes = findNodeForReinsertion(_node, _bvh);
  if (lRes.node == _bvh.root()) { return {false, 0, 0}; }
  assert(lRes.node != 0);

  uint32_t lBestPatchIndex = _bvh.patchIndex(lRes.node); // Check if node is already patched
  BVHNode *lBest           = nullptr;

  if (lBestPatchIndex == UINT32_MAX) {
    // Node is not patched ==> try to lock it
    IF_NOT_LOCK(lRes.node, _lockID) { return {false, 0, 0}; }
    lBest = _bvh.patchNode(lRes.node);
  } else {
    // Node is already owned by this thread ==> no need to lock it
    lBest = _bvh.getPatchedNode(lBestPatchIndex);
  }

  BVHNode *lNode           = _bvh[_node];
  BVHNode *lUnused         = _bvh[_unused];
  uint32_t lRootIndex      = lBest->parent;
  uint32_t lRootPatchIndex = _bvh.patchIndex(lRootIndex);
  BVHNode *lRoot           = nullptr;

  if (lRootPatchIndex == UINT32_MAX) {
    IF_NOT_LOCK(lRootIndex, _lockID) {
      RELEASE_LOCK_S(lRes.node, _lockID);
      return {false, 0, 0};
    }
    lRoot = _bvh.patchNode(lRootIndex);
  } else {
    lRoot = _bvh.getPatchedNode(lRootPatchIndex);
  }

  // Insert the unused node
  if (lBest->isLeftChild()) {
    lRoot->left     = _unused;
    lUnused->isLeft = TRUE;
  } else {
    lRoot->right    = _unused;
    lUnused->isLeft = FALSE;
  }


  // Insert the other nodes
  lUnused->parent = lRootIndex;
  lUnused->left   = lRes.node;
  lUnused->right  = _node;

  lBest->parent = _unused;
  lBest->isLeft = TRUE;
  lNode->parent = _unused;
  lNode->isLeft = FALSE;

  if (_update) {
    _bvh.nodeUpdated(lRes.node, lRes.level);
    _bvh.patchAABBFrom(_unused);
  }

  return {true, lRes.node, lRootIndex};
}




/*  ___  ___      _         _                        _       */
/*  |  \/  |     (_)       | |                      | |      */
/*  | .  . | __ _ _ _ __   | | _____ _ __ _ __   ___| |___   */
/*  | |\/| |/ _` | | '_ \  | |/ / _ \ '__| '_ \ / _ \ / __|  */
/*  | |  | | (_| | | | | | |   <  __/ |  | | | |  __/ \__ \  */
/*  \_|  |_/\__,_|_|_| |_| |_|\_\___|_|  |_| |_|\___|_|___/  */
/*                                                           */
/*                                                           */



__global__ void kFixTree(uint32_t *_leaf, SumMinCUDA _SMF, BVHNode *_nodes, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;

  AABB     lAABB;
  uint32_t lNode;
  uint32_t lLeft;
  uint32_t lRight;
  float    lSArea;

  for (uint32_t i = index; i < _num; i += stride) {
    lNode            = _leaf[i];
    _SMF.sums[lNode] = _nodes[lNode].surfaceArea;
    _SMF.mins[lNode] = _nodes[lNode].surfaceArea;
    lNode            = _nodes[lNode].parent;

    while (true) {
      uint32_t lOldLock = atomicCAS(&_SMF.flags[lNode], 0, 1);

      // Check if this thread is first. If yes break
      if (lOldLock == 0) { break; }

      lLeft  = _nodes[lNode].left;
      lRight = _nodes[lNode].right;
      lAABB  = _nodes[lLeft].bbox;
      lAABB.mergeWith(_nodes[lRight].bbox);
      lSArea = lAABB.surfaceArea();

      _nodes[lNode].bbox        = lAABB;
      _nodes[lNode].surfaceArea = lSArea;
      _nodes[lNode].numChildren = _nodes[lLeft].numChildren + _nodes[lRight].numChildren + 2;
      _SMF.sums[lNode]          = _SMF.sums[lLeft] + _SMF.sums[lRight] + lSArea;
      _SMF.mins[lNode]          = _SMF.mins[lLeft] < _SMF.mins[lRight] ? _SMF.mins[lLeft] : _SMF.mins[lRight];

      // Check if root
      if (lNode == _nodes[lNode].parent) { break; }
      lNode = _nodes[lNode].parent;
    }
  }
}


__global__ void kFixTree3_1(uint32_t *_toFix, SumMinCUDA _SMF, BVHNode *_nodes, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;

  for (uint32_t i = index; i < _num; i += stride) {
    uint32_t lNode = _toFix[i];
    if (lNode == UINT32_MAX) { continue; }

    while (true) {
      if (atomicAdd(&_SMF.flags[lNode], 1) != 0) { break; } // Stop when already locked (locked == 1)

      // Check if root
      if (lNode == _nodes[lNode].parent) { break; }
      lNode = _nodes[lNode].parent;
    }
  }
}

__global__ void kFixTree3_2(uint32_t *_toFix, SumMinCUDA _SMF, BVHNode *_nodes, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;

  AABB     lAABB;
  uint32_t lNode;
  uint32_t lLeft;
  uint32_t lRight;
  float    lSArea;

  for (uint32_t i = index; i < _num; i += stride) {
    lNode = _toFix[i];
    if (lNode == UINT32_MAX) { continue; }

    while (true) {
      if (atomicSub(&_SMF.flags[lNode], 1) != 1) { break; } // Stop when already locked (locked == 1)

      lLeft  = _nodes[lNode].left;
      lRight = _nodes[lNode].right;
      lAABB  = _nodes[lLeft].bbox;
      lAABB.mergeWith(_nodes[lRight].bbox);
      lSArea = lAABB.surfaceArea();

      _nodes[lNode].bbox        = lAABB;
      _nodes[lNode].surfaceArea = lSArea;
      _nodes[lNode].numChildren = _nodes[lLeft].numChildren + _nodes[lRight].numChildren + 2;
      _SMF.sums[lNode]          = _SMF.sums[lLeft] + _SMF.sums[lRight] + lSArea;
      _SMF.mins[lNode]          = _SMF.mins[lLeft] < _SMF.mins[lRight] ? _SMF.mins[lLeft] : _SMF.mins[lRight];

      // Check if root
      if (lNode == _nodes[lNode].parent) { break; }
      lNode = _nodes[lNode].parent;
    }
  }
}


__global__ void kCalcCost(float *_sum, float *_min, BVHNode *_BVHNode, float *_costOut, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;
  for (uint32_t i = index; i < _num; i += stride) {
    uint32_t lParent      = _BVHNode[i].parent;
    uint32_t lNumChildren = _BVHNode[i].numChildren;
    float    lSA          = _BVHNode[i].surfaceArea;
    bool     lCanRemove   = (lNumChildren != 0) && (i != lParent);

    _costOut[i] = lCanRemove ? ((lSA * lSA * lSA * (float)lNumChildren) / (_sum[i] * _min[i])) : 0.0f;
  }
}



__global__ void kRemoveAndReinsert(uint32_t *_todoList,
                                   PATCH *   _patches,
                                   uint32_t *_flags,
                                   uint32_t *_skip,
                                   uint32_t *_toFix,
                                   bool      _offsetAccess,
                                   bool      _retry,
                                   uint32_t  _chunk,
                                   uint32_t  _numChunks,
                                   uint32_t  _chunkSize) {
  uint32_t index   = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride  = blockDim.x * gridDim.x;
  uint32_t lLockID = index + 1;

  for (int32_t k = index; k < _chunkSize; k += stride) {
    uint32_t     lNodeIndex = _todoList[_offsetAccess ? k * _numChunks + _chunk : _chunk * _chunkSize + k];
    CUDA_RM_RES  lRmRes     = removeNode(lNodeIndex, _patches[k], _flags, lLockID);
    CUDA_INS_RES lR1, lR2;

    if (!lRmRes.res) {
      _patches[k].clear();

      if (_retry) {
        k -= stride;
        _retry = false;
        continue;
      }

      _skip[k] += 1;
      _toFix[k * 3 + 0] = UINT32_MAX;
      _toFix[k * 3 + 1] = UINT32_MAX;
      _toFix[k * 3 + 2] = UINT32_MAX;

      continue;
    }

    lR1 = reinsert(lRmRes.toInsert.n1, lRmRes.unused.n1, _patches[k], true, _flags, lLockID);
    lR2 = reinsert(lRmRes.toInsert.n2, lRmRes.unused.n2, _patches[k], false, _flags, lLockID);
    if (!lR1.res || !lR2.res) {
      _patches[k].clear();

      // Unlock Nodes
      RELEASE_LOCK_S(lRmRes.toInsert.n1, lLockID);
      RELEASE_LOCK_S(lRmRes.toInsert.n2, lLockID);
      RELEASE_LOCK_S(lRmRes.unused.n1, lLockID);
      RELEASE_LOCK_S(lRmRes.unused.n2, lLockID);
      RELEASE_LOCK_S(lRmRes.grandParentAndSibling.n1, lLockID);
      RELEASE_LOCK_S(lRmRes.grandParentAndSibling.n2, lLockID);
      if (lR1.res) {
        RELEASE_LOCK_S(lR1.best, lLockID);
        RELEASE_LOCK_S(lR1.root, lLockID);
      }
      if (lR2.res) {
        RELEASE_LOCK_S(lR2.best, lLockID);
        RELEASE_LOCK_S(lR2.root, lLockID);
      }

      _skip[k] += 1;
      _toFix[k * 3 + 0] = UINT32_MAX;
      _toFix[k * 3 + 1] = UINT32_MAX;
      _toFix[k * 3 + 2] = UINT32_MAX;

      continue;
    }

    _toFix[k * 3 + 0] = lRmRes.grandParentAndSibling.n1;
    _toFix[k * 3 + 1] = lRmRes.unused.n1;
    _toFix[k * 3 + 2] = lRmRes.unused.n2;
  }
}


__global__ void kApplyPatches(PATCH *_patches, uint32_t *_flags, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;
  for (uint32_t k = index; k < _num; k += stride) {
    if (_patches[k].empty()) { continue; }

    for (uint32_t l = 0; l < 10; ++l) {
      if (l >= _patches[k].size()) { break; }
      RELEASE_LOCK(_patches[k].getPatchedNodeIndex(l));
    }

    _patches[k].apply();
    _patches[k].clear();
  }
}



/*    ___  _                  _ _   _                  __                  _   _                   */
/*   / _ \| |                (_) | | |                / _|                | | (_)                  */
/*  / /_\ \ | __ _  ___  _ __ _| |_| |__  _ __ ___   | |_ _   _ _ __   ___| |_ _  ___  _ __  ___   */
/*  |  _  | |/ _` |/ _ \| '__| | __| '_ \| '_ ` _ \  |  _| | | | '_ \ / __| __| |/ _ \| '_ \/ __|  */
/*  | | | | | (_| | (_) | |  | | |_| | | | | | | | | | | | |_| | | | | (__| |_| | (_) | | | \__ \  */
/*  \_| |_/_|\__, |\___/|_|  |_|\__|_| |_|_| |_| |_| |_|  \__,_|_| |_|\___|\__|_|\___/|_| |_|___/  */
/*            __/ |                                                                                */
/*           |___/                                                                                 */


void fixTree(GPUWorkingMemory *_data, base::CUDAMemoryBVHPointer *_GPUbvh, uint32_t _blockSize) {
  if (!_data || !_GPUbvh) { return; }

  uint32_t lNumBlocks = (_data->numLeafNodes + _blockSize - 1) / _blockSize;
  kFixTree<<<lNumBlocks, _blockSize>>>(_data->leafNodes, _data->sumMin, _GPUbvh->nodes, _data->numLeafNodes);

  hipMemset(_data->sumMin.flags, 0, _data->sumMin.num * sizeof(uint32_t));
}

void fixTree3(GPUWorkingMemory *_data, BVHTest::base::CUDAMemoryBVHPointer *_GPUbvh, uint32_t _blockSize) {
  if (!_data || !_GPUbvh) { return; }

  uint32_t lNumBlocks = (_data->numNodesToFix + _blockSize - 1) / _blockSize;
  kFixTree3_1<<<lNumBlocks, _blockSize>>>(_data->nodesToFix, _data->sumMin, _GPUbvh->nodes, _data->numNodesToFix);
  kFixTree3_2<<<lNumBlocks, _blockSize>>>(_data->nodesToFix, _data->sumMin, _GPUbvh->nodes, _data->numNodesToFix);
}



void doAlgorithmStep(GPUWorkingMemory *    _data,
                     CUDAMemoryBVHPointer *_GPUbvh,
                     uint32_t              _numChunks,
                     uint32_t              _chunkSize,
                     uint32_t              _blockSize,
                     bool                  _offsetAccess,
                     bool                  _retry) {
  if (!_data || !_GPUbvh) { return; }

  hipError_t lRes;
  uint32_t    lNumBlocksAll   = (_data->sumMin.num + _blockSize - 1) / _blockSize;
  uint32_t    lNumBlocksChunk = (_chunkSize + _blockSize - 1) / _blockSize;

  kCalcCost<<<lNumBlocksAll, _blockSize>>>(
      _data->sumMin.sums, _data->sumMin.mins, _GPUbvh->nodes, _data->todoNodes.costs, _data->sumMin.num);

  CUDA_RUN(hipcub::DeviceRadixSort::SortPairsDescending(_data->cubSortTempStorage,
                                                     _data->cubSortTempStorageSize,
                                                     _data->todoNodes.costs,
                                                     _data->todoSorted.costs,
                                                     _data->todoNodes.nodes,
                                                     _data->todoSorted.nodes,
                                                     _data->todoNodes.num));

  for (uint32_t i = 0; i < _numChunks; ++i) {
    kRemoveAndReinsert<<<lNumBlocksChunk, _blockSize>>>(_data->todoSorted.nodes,
                                                        _data->patches,
                                                        _data->sumMin.flags,
                                                        _data->skipped,
                                                        _data->nodesToFix,
                                                        _offsetAccess,
                                                        _retry,
                                                        i,
                                                        _numChunks,
                                                        _chunkSize);

    kApplyPatches<<<lNumBlocksChunk, _blockSize>>>(_data->patches, _data->sumMin.flags, _chunkSize);

    fixTree3(_data, _GPUbvh, _blockSize);
  }

error:
  return;
}


uint32_t calcNumSkipped(GPUWorkingMemory *_data) {
  hipError_t lRes;
  uint32_t    lSkipped    = 0;
  uint32_t *  lDevSkipped = nullptr;

  ALLOCATE(&lDevSkipped, 1, uint32_t);

  CUDA_RUN(hipcub::DeviceReduce::Sum(
      _data->cubSortTempStorage, _data->cubSortTempStorageSize, _data->skipped, lDevSkipped, _data->numSkipped));

  CUDA_RUN(hipMemcpy(&lSkipped, lDevSkipped, sizeof(uint32_t), hipMemcpyDeviceToHost));

error:
  FREE2(lDevSkipped);
  return lSkipped;
}


void doCudaDevSync() { hipDeviceSynchronize(); }

/*  ___  ___                                                                                              _     */
/*  |  \/  |                                                                                             | |    */
/*  | .  . | ___ _ __ ___   ___  _ __ _   _   _ __ ___   __ _ _ __   __ _  __ _  ___ _ __ ___   ___ _ __ | |_   */
/*  | |\/| |/ _ \ '_ ` _ \ / _ \| '__| | | | | '_ ` _ \ / _` | '_ \ / _` |/ _` |/ _ \ '_ ` _ \ / _ \ '_ \| __|  */
/*  | |  | |  __/ | | | | | (_) | |  | |_| | | | | | | | (_| | | | | (_| | (_| |  __/ | | | | |  __/ | | | |_   */
/*  \_|  |_/\___|_| |_| |_|\___/|_|   \__, | |_| |_| |_|\__,_|_| |_|\__,_|\__, |\___|_| |_| |_|\___|_| |_|\__|  */
/*                                     __/ |                               __/ |                                */
/*                                    |___/                               |___/                                 */


GPUWorkingMemory allocateMemory(CUDAMemoryBVHPointer *_bvh, uint32_t _batchSize, uint32_t _numFaces) {
  hipProfilerStart();
  GPUWorkingMemory lMem;

  lMem.result = true;
  hipError_t lRes;
  size_t      lCubTempStorage1 = 0;
  size_t      lCubTempStorage2 = 0;
  uint32_t *  lTemp            = nullptr;

  lMem.sumMin.num     = _bvh->numNodes;
  lMem.todoNodes.num  = _bvh->numNodes;
  lMem.todoSorted.num = _bvh->numNodes;
  lMem.numLeafNodes   = _numFaces;
  lMem.numPatches     = _batchSize;
  lMem.numSkipped     = _batchSize;
  lMem.numNodesToFix  = _batchSize * 3;

  ALLOCATE(&lMem.sumMin.sums, lMem.sumMin.num, float);
  ALLOCATE(&lMem.sumMin.mins, lMem.sumMin.num, float);
  ALLOCATE(&lMem.sumMin.flags, lMem.sumMin.num, uint32_t);
  ALLOCATE(&lMem.todoNodes.nodes, lMem.todoNodes.num, uint32_t);
  ALLOCATE(&lMem.todoNodes.costs, lMem.todoNodes.num, float);
  ALLOCATE(&lMem.todoSorted.nodes, lMem.todoSorted.num, uint32_t);
  ALLOCATE(&lMem.todoSorted.costs, lMem.todoSorted.num, float);
  ALLOCATE(&lMem.leafNodes, lMem.numLeafNodes, uint32_t);
  ALLOCATE(&lMem.patches, lMem.numPatches, PATCH);
  ALLOCATE(&lMem.skipped, lMem.numSkipped, uint32_t);
  ALLOCATE(&lMem.nodesToFix, lMem.numNodesToFix, uint32_t);



  // This only calculates the memory requirements
  CUDA_RUN(hipcub::DeviceRadixSort::SortPairsDescending(lMem.cubSortTempStorage,
                                                     lCubTempStorage1,
                                                     lMem.todoNodes.costs,
                                                     lMem.todoSorted.costs,
                                                     lMem.todoNodes.nodes,
                                                     lMem.todoSorted.nodes,
                                                     lMem.todoNodes.num));

  CUDA_RUN(hipcub::DeviceReduce::Sum(lMem.cubSortTempStorage, lCubTempStorage2, lMem.skipped, lTemp, lMem.numSkipped));

  lMem.cubSortTempStorageSize = lCubTempStorage1 > lCubTempStorage2 ? lCubTempStorage1 : lCubTempStorage2;

  ALLOCATE(&lMem.cubSortTempStorage, lMem.cubSortTempStorageSize, uint8_t);

  return lMem;

error:
  lMem.result = false;

  FREE(lMem.sumMin.sums, lMem.sumMin.num);
  FREE(lMem.sumMin.mins, lMem.sumMin.num);
  FREE(lMem.sumMin.flags, lMem.sumMin.num);
  FREE(lMem.todoNodes.nodes, lMem.todoNodes.num);
  FREE(lMem.todoNodes.costs, lMem.todoNodes.num);
  FREE(lMem.todoSorted.nodes, lMem.todoNodes.num);
  FREE(lMem.todoSorted.costs, lMem.todoNodes.num);
  FREE(lMem.leafNodes, lMem.numLeafNodes);
  FREE(lMem.patches, lMem.numPatches);
  FREE(lMem.skipped, lMem.numSkipped);
  FREE(lMem.nodesToFix, lMem.numNodesToFix);
  FREE(lMem.cubSortTempStorage, lMem.cubSortTempStorageSize);

  return lMem;
}

void freeMemory(GPUWorkingMemory *_data) {
  _data->result = false;

  FREE(_data->sumMin.sums, _data->sumMin.num);
  FREE(_data->sumMin.mins, _data->sumMin.num);
  FREE(_data->sumMin.flags, _data->sumMin.num);
  FREE(_data->todoNodes.nodes, _data->todoNodes.num);
  FREE(_data->todoNodes.costs, _data->todoNodes.num);
  FREE(_data->todoSorted.nodes, _data->todoSorted.num);
  FREE(_data->todoSorted.costs, _data->todoSorted.num);
  FREE(_data->leafNodes, _data->numLeafNodes);
  FREE(_data->patches, _data->numPatches);
  FREE(_data->skipped, _data->numSkipped);
  FREE(_data->nodesToFix, _data->numNodesToFix);
  FREE(_data->cubSortTempStorage, _data->cubSortTempStorageSize);
  hipProfilerStop();
}


void initData(GPUWorkingMemory *_data, CUDAMemoryBVHPointer *_GPUbvh, uint32_t _blockSize) {
  if (!_data || !_GPUbvh) { return; }

  hipError_t   lRes;
  uint32_t      lNumBlocksAll     = (_data->todoNodes.num + _blockSize - 1) / _blockSize;
  uint32_t      lNumBlocksPatches = (_data->numPatches + _blockSize - 1) / _blockSize;
  void *        lTempStorage      = nullptr;
  int *         lNumSelected      = nullptr;
  size_t        lTempStorageSize  = 0;
  CUBLeafSelect lSelector(_GPUbvh->nodes);

  kResetTodoData<<<lNumBlocksAll, _blockSize>>>(_data->todoNodes.nodes, _data->todoNodes.num);
  kInitPatches<<<lNumBlocksPatches, _blockSize>>>(_data->patches, _GPUbvh->bvh, _data->numPatches);

  CUDA_RUN(hipMemset(_data->sumMin.flags, 0, _data->sumMin.num * sizeof(uint32_t)));
  CUDA_RUN(hipMemset(_data->skipped, 0, _data->numSkipped * sizeof(uint32_t)));

  ALLOCATE(&lNumSelected, 1, int);

  CUDA_RUN(hipcub::DeviceSelect::If(lTempStorage,
                                 lTempStorageSize,
                                 _data->todoNodes.nodes,
                                 _data->leafNodes,
                                 lNumSelected,
                                 _data->todoNodes.num,
                                 lSelector));

  ALLOCATE(&lTempStorage, lTempStorageSize, uint8_t);

  CUDA_RUN(hipcub::DeviceSelect::If(lTempStorage,
                                 lTempStorageSize,
                                 _data->todoNodes.nodes,
                                 _data->leafNodes,
                                 lNumSelected,
                                 _data->todoNodes.num,
                                 lSelector));

error:
  hipFree(lNumSelected);
  FREE(lTempStorage, lTempStorageSize);
}
