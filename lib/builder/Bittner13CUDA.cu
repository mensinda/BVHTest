#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2018 Daniel Mensinger
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda/cudaFN.hpp"
#include "Bittner13CUDA.hpp"
#include <hipcub/hipcub.hpp>
#include <iostream>

using namespace glm;
using namespace std;
using namespace BVHTest;
using namespace BVHTest::base;
using namespace BVHTest::cuda;

#define CUDA_RUN(call)                                                                                                 \
  lRes = call;                                                                                                         \
  if (lRes != hipSuccess) {                                                                                           \
    cout << "CUDA ERROR (" << __FILE__ << ":" << __LINE__ << "): " << hipGetErrorString(lRes) << endl;                \
    goto error;                                                                                                        \
  }

#define ALLOCATE(ptr, num, type) CUDA_RUN(hipMalloc(ptr, num * sizeof(type)));
#define FREE(ptr, num)                                                                                                 \
  hipFree(ptr);                                                                                                       \
  ptr = nullptr;                                                                                                       \
  num = 0;

struct CUBLeafSelect {
  BVHNode *nodes;

  HIPCUB_RUNTIME_FUNCTION __forceinline__ CUBLeafSelect(BVHNode *_n) : nodes(_n) {}

  __device__ __forceinline__ bool operator()(const uint32_t &a) const { return nodes[a].numChildren == 0; }
};

__global__ void kResetTodoData(uint32_t *_nodes, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;
  for (uint32_t i = index; i < _num; i += stride) { _nodes[i] = i; }
}

__global__ void kResetLocks(uint32_t *_locks, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;
  for (uint32_t i = index; i < _num; i += stride) { _locks[i] = 0; }
}


/*  ___  ___      _         _                        _       */
/*  |  \/  |     (_)       | |                      | |      */
/*  | .  . | __ _ _ _ __   | | _____ _ __ _ __   ___| |___   */
/*  | |\/| |/ _` | | '_ \  | |/ / _ \ '__| '_ \ / _ \ / __|  */
/*  | |  | | (_| | | | | | |   <  __/ |  | | | |  __/ \__ \  */
/*  \_|  |_/\__,_|_|_| |_| |_|\_\___|_|  |_| |_|\___|_|___/  */
/*                                                           */
/*                                                           */



__global__ void kFixTree(uint32_t *_leaf, float *_sum, float *_min, BVHNode *_node, uint32_t *_flag, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;

  AABB     lAABB;
  uint32_t lNode;
  uint32_t lLeft;
  uint32_t lRight;
  float    lSArea;

  for (uint32_t i = index; i < _num; i += stride) {
    lNode       = _leaf[i];
    _sum[lNode] = _node[lNode].surfaceArea;
    _min[lNode] = _node[lNode].surfaceArea;
    lNode       = _node[lNode].parent;

    while (true) {
      uint32_t lOldLock = atomicCAS(&_flag[lNode], 0, 1);

      // Check if this thread is first. If yes break
      if (lOldLock == 0) { break; }

      lLeft  = _node[lNode].left;
      lRight = _node[lNode].right;
      lAABB  = _node[lLeft].bbox;
      lAABB.mergeWith(_node[lRight].bbox);
      lSArea = lAABB.surfaceArea();

      _node[lNode].bbox        = lAABB;
      _node[lNode].surfaceArea = lSArea;
      _node[lNode].numChildren = _node[lLeft].numChildren + _node[lRight].numChildren + 2;
      _sum[lNode]              = _sum[lLeft] + _sum[lRight] + lSArea;
      _min[lNode]              = _min[lLeft] < _min[lRight] ? _min[lLeft] : _min[lRight];

      // Check if root
      if (lNode == _node[lNode].parent) { break; }
      lNode = _node[lNode].parent;
    }
  }
}


__global__ void kCalcCost(float *_sum, float *_min, BVHNode *_BVHNode, float *_costOut, uint32_t _num) {
  uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t stride = blockDim.x * gridDim.x;
  for (uint32_t i = index; i < _num; i += stride) {
    uint32_t lParent      = _BVHNode[i].parent;
    uint32_t lNumChildren = _BVHNode[i].numChildren;
    float    lSA          = _BVHNode[i].surfaceArea;
    bool     lCanRemove   = (lNumChildren != 0) && (i != lParent) && (lParent != _BVHNode[lParent].parent);

    _costOut[i] = lCanRemove ? ((lSA * lSA * lSA * (float)lNumChildren) / (_sum[i] * _min[i])) : 0.0f;
  }
}


/*  ___  ___                                                                                              _     */
/*  |  \/  |                                                                                             | |    */
/*  | .  . | ___ _ __ ___   ___  _ __ _   _   _ __ ___   __ _ _ __   __ _  __ _  ___ _ __ ___   ___ _ __ | |_   */
/*  | |\/| |/ _ \ '_ ` _ \ / _ \| '__| | | | | '_ ` _ \ / _` | '_ \ / _` |/ _` |/ _ \ '_ ` _ \ / _ \ '_ \| __|  */
/*  | |  | |  __/ | | | | | (_) | |  | |_| | | | | | | | (_| | | | | (_| | (_| |  __/ | | | | |  __/ | | | |_   */
/*  \_|  |_/\___|_| |_| |_|\___/|_|   \__, | |_| |_| |_|\__,_|_| |_|\__,_|\__, |\___|_| |_| |_|\___|_| |_|\__|  */
/*                                     __/ |                               __/ |                                */
/*                                    |___/                               |___/                                 */


GPUWorkingMemory allocateMemory(CUDAMemoryBVHPointer *_bvh, uint32_t _batchSize, uint32_t _numFaces) {
  GPUWorkingMemory lMem;

  lMem.result = true;
  hipError_t lRes;

  lMem.sumMin.num     = _bvh->numNodes;
  lMem.todoNodes.num  = _bvh->numNodes;
  lMem.todoSorted.num = _bvh->numNodes;
  lMem.numLeafNodes   = _numFaces;
  lMem.numPatches     = _batchSize;

  ALLOCATE(&lMem.sumMin.sums, lMem.sumMin.num, float);
  ALLOCATE(&lMem.sumMin.mins, lMem.sumMin.num, float);
  ALLOCATE(&lMem.sumMin.flags, lMem.sumMin.num, uint32_t);
  ALLOCATE(&lMem.todoNodes.nodes, lMem.todoNodes.num, uint32_t);
  ALLOCATE(&lMem.todoNodes.costs, lMem.todoNodes.num, float);
  ALLOCATE(&lMem.todoSorted.nodes, lMem.todoSorted.num, uint32_t);
  ALLOCATE(&lMem.todoSorted.costs, lMem.todoSorted.num, float);
  ALLOCATE(&lMem.leafNodes, lMem.numLeafNodes, uint32_t);
  ALLOCATE(&lMem.patches, lMem.numPatches, PATCH);

  // This only calculates the memory requirements
  CUDA_RUN(hipcub::DeviceRadixSort::SortPairsDescending(lMem.cubSortTempStorage,
                                                     lMem.cubSortTempStorageSize,
                                                     lMem.todoNodes.costs,
                                                     lMem.todoSorted.costs,
                                                     lMem.todoNodes.nodes,
                                                     lMem.todoSorted.nodes,
                                                     lMem.todoNodes.num));

  ALLOCATE(&lMem.cubSortTempStorage, lMem.cubSortTempStorageSize, uint8_t);

  return lMem;

error:
  lMem.result = false;

  FREE(lMem.sumMin.sums, lMem.sumMin.num);
  FREE(lMem.sumMin.mins, lMem.sumMin.num);
  FREE(lMem.sumMin.flags, lMem.sumMin.num);
  FREE(lMem.todoNodes.nodes, lMem.todoNodes.num);
  FREE(lMem.todoNodes.costs, lMem.todoNodes.num);
  FREE(lMem.todoSorted.nodes, lMem.todoNodes.num);
  FREE(lMem.todoSorted.costs, lMem.todoNodes.num);
  FREE(lMem.leafNodes, lMem.numLeafNodes);
  FREE(lMem.patches, lMem.numPatches);
  FREE(lMem.cubSortTempStorage, lMem.cubSortTempStorageSize);

  return lMem;
}

void freeMemory(GPUWorkingMemory *_data) {
  _data->result = false;

  FREE(_data->sumMin.sums, _data->sumMin.num);
  FREE(_data->sumMin.mins, _data->sumMin.num);
  FREE(_data->sumMin.flags, _data->sumMin.num);
  FREE(_data->todoNodes.nodes, _data->todoNodes.num);
  FREE(_data->todoNodes.costs, _data->todoNodes.num);
  FREE(_data->todoSorted.nodes, _data->todoSorted.num);
  FREE(_data->todoSorted.costs, _data->todoSorted.num);
  FREE(_data->leafNodes, _data->numLeafNodes);
  FREE(_data->patches, _data->numPatches);
  FREE(_data->cubSortTempStorage, _data->cubSortTempStorageSize);
}


void initData(GPUWorkingMemory *_data, CUDAMemoryBVHPointer *_GPUbvh, uint32_t _blockSize) {
  if (!_data || !_GPUbvh) { return; }

  uint32_t lNumBlocks = (_data->todoNodes.num + _blockSize - 1) / _blockSize;
  kResetTodoData<<<lNumBlocks, _blockSize>>>(_data->todoNodes.nodes, _data->todoNodes.num);

  resetLocks(_data, _blockSize);

  hipError_t   lRes;
  CUBLeafSelect lSelector(_GPUbvh->nodes);
  void *        lTempStorage     = nullptr;
  int *         lNumSelected     = nullptr;
  size_t        lTempStorageSize = 0;

  ALLOCATE(&lNumSelected, 1, int);

  CUDA_RUN(hipcub::DeviceSelect::If(lTempStorage,
                                 lTempStorageSize,
                                 _data->todoNodes.nodes,
                                 _data->leafNodes,
                                 lNumSelected,
                                 _data->todoNodes.num,
                                 lSelector));

  ALLOCATE(&lTempStorage, lTempStorageSize, uint8_t);

  CUDA_RUN(hipcub::DeviceSelect::If(lTempStorage,
                                 lTempStorageSize,
                                 _data->todoNodes.nodes,
                                 _data->leafNodes,
                                 lNumSelected,
                                 _data->todoNodes.num,
                                 lSelector));

error:
  hipFree(lNumSelected);
  FREE(lTempStorage, lTempStorageSize);
}


void resetLocks(GPUWorkingMemory *_data, uint32_t _blockSize) {
  if (!_data) { return; }

  uint32_t lNumBlocks = (_data->sumMin.num + _blockSize - 1) / _blockSize;
  kResetLocks<<<lNumBlocks, _blockSize>>>(_data->sumMin.flags, _data->sumMin.num);
}


/*    ___  _                  _ _   _                  __                  _   _                   */
/*   / _ \| |                (_) | | |                / _|                | | (_)                  */
/*  / /_\ \ | __ _  ___  _ __ _| |_| |__  _ __ ___   | |_ _   _ _ __   ___| |_ _  ___  _ __  ___   */
/*  |  _  | |/ _` |/ _ \| '__| | __| '_ \| '_ ` _ \  |  _| | | | '_ \ / __| __| |/ _ \| '_ \/ __|  */
/*  | | | | | (_| | (_) | |  | | |_| | | | | | | | | | | | |_| | | | | (__| |_| | (_) | | | \__ \  */
/*  \_| |_/_|\__, |\___/|_|  |_|\__|_| |_|_| |_| |_| |_|  \__,_|_| |_|\___|\__|_|\___/|_| |_|___/  */
/*            __/ |                                                                                */
/*           |___/                                                                                 */


void fixTree(GPUWorkingMemory *_data, base::CUDAMemoryBVHPointer *_GPUbvh, uint32_t _blockSize) {
  if (!_data || !_GPUbvh) { return; }

  uint32_t lNumBlocks = (_data->numLeafNodes + _blockSize - 1) / _blockSize;
  kFixTree<<<lNumBlocks, _blockSize>>>(_data->leafNodes,
                                       _data->sumMin.sums,
                                       _data->sumMin.mins,
                                       _GPUbvh->nodes,
                                       _data->sumMin.flags,
                                       _data->numLeafNodes);

  resetLocks(_data, _blockSize);
}

void calculateCost(GPUWorkingMemory *_data, base::CUDAMemoryBVHPointer *_GPUbvh, uint32_t _blockSize) {
  if (!_data || !_GPUbvh) { return; }

  hipError_t lRes;
  uint32_t    lNumBlocks = (_data->sumMin.num + _blockSize - 1) / _blockSize;
  kCalcCost<<<lNumBlocks, _blockSize>>>(
      _data->sumMin.sums, _data->sumMin.mins, _GPUbvh->nodes, _data->todoNodes.costs, _data->sumMin.num);

  CUDA_RUN(hipcub::DeviceRadixSort::SortPairsDescending(_data->cubSortTempStorage,
                                                     _data->cubSortTempStorageSize,
                                                     _data->todoNodes.costs,
                                                     _data->todoSorted.costs,
                                                     _data->todoNodes.nodes,
                                                     _data->todoSorted.nodes,
                                                     _data->todoNodes.num));

error:
  return;
}
