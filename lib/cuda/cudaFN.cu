#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2018 Daniel Mensinger
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudaFN.hpp"
#include <iostream>

using namespace glm;
using namespace std;
using namespace BVHTest;
using namespace BVHTest::base;
using namespace BVHTest::cuda;

#define CUDA_RUN(call)                                                                                                 \
  lRes = call;                                                                                                         \
  if (lRes != hipSuccess) { goto error; }

extern "C" bool copyBVHToGPU(BVH *_bvh, CUDAMemoryBVHPointer *_ptr) {
  if (!_bvh || !_ptr) { return false; }

  size_t      lSize = _bvh->size() * sizeof(BVHNode);
  BVH         lTempBVH;
  hipError_t lRes;

  _ptr->numNodes = _bvh->size();

  CUDA_RUN(hipMalloc(&_ptr->nodes, lSize));
  CUDA_RUN(hipMemcpy(_ptr->nodes, _bvh->data(), lSize, hipMemcpyHostToDevice));

  lTempBVH.setNewRoot(_bvh->root());
  lTempBVH.setMaxLevel(_bvh->maxLevel());
  lTempBVH.setMemory(_ptr->nodes, _bvh->size(), _bvh->size());

  CUDA_RUN(hipMalloc(&_ptr->bvh, sizeof(BVH)));
  CUDA_RUN(hipMemcpy(_ptr->bvh, &lTempBVH, sizeof(BVH), hipMemcpyHostToDevice));

  lTempBVH.setMemory(nullptr, 0, 0); // Avoid destructor segfault

  return true;

error:
  cout << "CUDA ERROR: " << hipGetErrorString(lRes) << endl;

  hipFree(_ptr->nodes);
  hipFree(_ptr->bvh);

  lTempBVH.setMemory(nullptr, 0, 0); // Avoid destructor segfault

  _ptr->bvh      = nullptr;
  _ptr->nodes    = nullptr;
  _ptr->numNodes = 0;
  return false;
}

extern "C" bool copyMeshToGPU(Mesh *_mesh, MeshRaw *_meshOut) {
  if (!_mesh) { return false; }

  hipError_t lRes;
  uint32_t    lVertSize  = _mesh->vert.size() * sizeof(vec3);
  uint32_t    lNromSize  = _mesh->norm.size() * sizeof(vec3);
  uint32_t    lFacesSize = _mesh->faces.size() * sizeof(Triangle);

  CUDA_RUN(hipMalloc(&_meshOut->vert, lVertSize));
  CUDA_RUN(hipMalloc(&_meshOut->norm, lNromSize));
  CUDA_RUN(hipMalloc(&_meshOut->faces, lFacesSize));

  CUDA_RUN(hipMemcpy(_meshOut->vert, _mesh->vert.data(), lVertSize, hipMemcpyHostToDevice));
  CUDA_RUN(hipMemcpy(_meshOut->norm, _mesh->norm.data(), lNromSize, hipMemcpyHostToDevice));
  CUDA_RUN(hipMemcpy(_meshOut->faces, _mesh->faces.data(), lFacesSize, hipMemcpyHostToDevice));

  _meshOut->numVert  = _mesh->vert.size();
  _meshOut->numNorm  = _mesh->norm.size();
  _meshOut->numFaces = _mesh->faces.size();

  return true;

error:
  cout << "CUDA ERROR: " << hipGetErrorString(lRes) << endl;

  if (_meshOut->vert) { hipFree(_meshOut->vert); }
  if (_meshOut->norm) { hipFree(_meshOut->norm); }
  if (_meshOut->faces) { hipFree(_meshOut->faces); }

  _meshOut->vert  = nullptr;
  _meshOut->norm  = nullptr;
  _meshOut->faces = nullptr;
  return false;
}



extern "C" bool copyBVHToHost(CUDAMemoryBVHPointer *_bvh, base::BVH *_ptr) {
  if (!_bvh->bvh || !_bvh->nodes || !_ptr) { return false; }

  hipError_t lRes;
  BVH         lTempBVH;
  uint32_t    lSize   = 0;
  BVHNode *   lSource = nullptr;

  CUDA_RUN(hipMemcpy(&lTempBVH, _bvh->bvh, sizeof(BVH), hipMemcpyDeviceToHost));

  _ptr->setNewRoot(lTempBVH.root());
  _ptr->setMaxLevel(lTempBVH.maxLevel());
  _ptr->resize(lTempBVH.size());

  lSize       = lTempBVH.size() * sizeof(BVHNode);
  _bvh->nodes = lTempBVH.data();

  CUDA_RUN(hipMemcpy(_ptr->data(), _bvh->nodes, lSize, hipMemcpyDeviceToHost));

error:
  if (lRes != hipSuccess) { cout << "CUDA ERROR: " << hipGetErrorString(lRes) << endl; }

  hipFree(_bvh->nodes);
  hipFree(_bvh->bvh);

  lTempBVH.setMemory(nullptr, 0, 0); // Avoid destructor segfault

  _bvh->bvh      = nullptr;
  _bvh->nodes    = nullptr;
  _bvh->numNodes = 0;

  return lRes == hipSuccess;
}


extern "C" bool copyMeshToHost(base::MeshRaw *_mesh, base::Mesh *_meshOut) {
  if (!_mesh || !_mesh->vert || !_mesh->norm || !_mesh->faces || !_meshOut) { return false; }

  hipError_t lRes;

  _meshOut->vert.resize(_mesh->numVert);
  _meshOut->norm.resize(_mesh->numNorm);
  _meshOut->faces.resize(_mesh->numFaces);

  uint32_t lVertSize  = _mesh->numVert * sizeof(vec3);
  uint32_t lNromSize  = _mesh->numNorm * sizeof(vec3);
  uint32_t lFacesSize = _mesh->numFaces * sizeof(Triangle);

  CUDA_RUN(hipMemcpy(_meshOut->vert.data(), _mesh->vert, lVertSize, hipMemcpyDeviceToHost));
  CUDA_RUN(hipMemcpy(_meshOut->norm.data(), _mesh->norm, lNromSize, hipMemcpyDeviceToHost));
  CUDA_RUN(hipMemcpy(_meshOut->faces.data(), _mesh->faces, lFacesSize, hipMemcpyDeviceToHost));

error:
  if (lRes != hipSuccess) { cout << "CUDA ERROR: " << hipGetErrorString(lRes) << endl; }

  hipFree(_mesh->vert);
  hipFree(_mesh->norm);
  hipFree(_mesh->faces);

  _mesh->vert  = nullptr;
  _mesh->norm  = nullptr;
  _mesh->faces = nullptr;

  _mesh->numVert  = 0;
  _mesh->numNorm  = 0;
  _mesh->numFaces = 0;

  return lRes == hipSuccess;
}
