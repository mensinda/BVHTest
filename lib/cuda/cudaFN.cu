#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2018 Daniel Mensinger
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudaFN.hpp"
#include "bucketSelect.cu"
#include <iostream>

using namespace glm;
using namespace std;
using namespace BVHTest;
using namespace BVHTest::base;
using namespace BVHTest::cuda;
using namespace BucketSelect;

#define CUDA_RUN(call)                                                                                                 \
  lRes = call;                                                                                                         \
  if (lRes != hipSuccess) { goto error; }

extern "C" bool copyBVHToGPU(BVH *_bvh, CUDAMemoryBVHPointer *_ptr) {
  if (!_bvh || !_ptr) { return false; }

  if (_bvh->size() == 0) {
    _ptr->bvh      = nullptr;
    _ptr->nodes    = nullptr;
    _ptr->numNodes = 0;
    return true;
  }

  size_t      lSize = _bvh->size() * sizeof(BVHNode);
  BVH         lTempBVH;
  hipError_t lRes;

  _ptr->numNodes = _bvh->size();

  CUDA_RUN(hipMalloc(&_ptr->nodes, lSize));
  CUDA_RUN(hipMemcpy(_ptr->nodes, _bvh->data(), lSize, hipMemcpyHostToDevice));

  lTempBVH.setNewRoot(_bvh->root());
  lTempBVH.setMaxLevel(_bvh->maxLevel());
  lTempBVH.setMemory(_ptr->nodes, _bvh->size(), _bvh->size());

  CUDA_RUN(hipMalloc(&_ptr->bvh, sizeof(BVH)));
  CUDA_RUN(hipMemcpy(_ptr->bvh, &lTempBVH, sizeof(BVH), hipMemcpyHostToDevice));

  lTempBVH.setMemory(nullptr, 0, 0); // Avoid destructor segfault

  return true;

error:
  cout << "CUDA ERROR: " << hipGetErrorString(lRes) << endl;

  hipFree(_ptr->nodes);
  hipFree(_ptr->bvh);

  lTempBVH.setMemory(nullptr, 0, 0); // Avoid destructor segfault

  _ptr->bvh      = nullptr;
  _ptr->nodes    = nullptr;
  _ptr->numNodes = 0;
  return false;
}

extern "C" bool copyMeshToGPU(Mesh *_mesh, MeshRaw *_meshOut) {
  if (!_mesh) { return false; }

  hipError_t lRes;
  uint32_t    lVertSize  = _mesh->vert.size() * sizeof(vec3);
  uint32_t    lNromSize  = _mesh->norm.size() * sizeof(vec3);
  uint32_t    lFacesSize = _mesh->faces.size() * sizeof(Triangle);

  CUDA_RUN(hipMalloc(&_meshOut->vert, lVertSize));
  CUDA_RUN(hipMalloc(&_meshOut->norm, lNromSize));
  CUDA_RUN(hipMalloc(&_meshOut->faces, lFacesSize));

  CUDA_RUN(hipMemcpy(_meshOut->vert, _mesh->vert.data(), lVertSize, hipMemcpyHostToDevice));
  CUDA_RUN(hipMemcpy(_meshOut->norm, _mesh->norm.data(), lNromSize, hipMemcpyHostToDevice));
  CUDA_RUN(hipMemcpy(_meshOut->faces, _mesh->faces.data(), lFacesSize, hipMemcpyHostToDevice));

  _meshOut->numVert  = _mesh->vert.size();
  _meshOut->numNorm  = _mesh->norm.size();
  _meshOut->numFaces = _mesh->faces.size();

  return true;

error:
  cout << "CUDA ERROR: " << hipGetErrorString(lRes) << endl;

  if (_meshOut->vert) { hipFree(_meshOut->vert); }
  if (_meshOut->norm) { hipFree(_meshOut->norm); }
  if (_meshOut->faces) { hipFree(_meshOut->faces); }

  _meshOut->vert  = nullptr;
  _meshOut->norm  = nullptr;
  _meshOut->faces = nullptr;
  return false;
}



extern "C" bool copyBVHToHost(CUDAMemoryBVHPointer *_bvh, base::BVH *_ptr) {
  if (!_bvh->bvh || !_bvh->nodes || !_ptr) { return false; }

  if (_bvh->numNodes == 0) { return true; }

  hipError_t lRes;
  BVH         lTempBVH;
  uint32_t    lSize = 0;

  CUDA_RUN(hipMemcpy(&lTempBVH, _bvh->bvh, sizeof(BVH), hipMemcpyDeviceToHost));

  _ptr->setNewRoot(lTempBVH.root());
  _ptr->setMaxLevel(lTempBVH.maxLevel());
  _ptr->resize(lTempBVH.size());

  lSize       = lTempBVH.size() * sizeof(BVHNode);
  _bvh->nodes = lTempBVH.data();

  CUDA_RUN(hipMemcpy(_ptr->data(), _bvh->nodes, lSize, hipMemcpyDeviceToHost));

error:
  if (lRes != hipSuccess) { cout << "CUDA ERROR: " << hipGetErrorString(lRes) << endl; }

  hipFree(_bvh->nodes);
  hipFree(_bvh->bvh);

  lTempBVH.setMemory(nullptr, 0, 0); // Avoid destructor segfault

  _bvh->bvh      = nullptr;
  _bvh->nodes    = nullptr;
  _bvh->numNodes = 0;

  return lRes == hipSuccess;
}


extern "C" bool copyMeshToHost(base::MeshRaw *_mesh, base::Mesh *_meshOut) {
  if (!_mesh || !_mesh->vert || !_mesh->norm || !_mesh->faces || !_meshOut) { return false; }

  hipError_t lRes;

  _meshOut->vert.resize(_mesh->numVert);
  _meshOut->norm.resize(_mesh->numNorm);
  _meshOut->faces.resize(_mesh->numFaces);

  uint32_t lVertSize  = _mesh->numVert * sizeof(vec3);
  uint32_t lNromSize  = _mesh->numNorm * sizeof(vec3);
  uint32_t lFacesSize = _mesh->numFaces * sizeof(Triangle);

  CUDA_RUN(hipMemcpy(_meshOut->vert.data(), _mesh->vert, lVertSize, hipMemcpyDeviceToHost));
  CUDA_RUN(hipMemcpy(_meshOut->norm.data(), _mesh->norm, lNromSize, hipMemcpyDeviceToHost));
  CUDA_RUN(hipMemcpy(_meshOut->faces.data(), _mesh->faces, lFacesSize, hipMemcpyDeviceToHost));

error:
  if (lRes != hipSuccess) { cout << "CUDA ERROR: " << hipGetErrorString(lRes) << endl; }

  hipFree(_mesh->vert);
  hipFree(_mesh->norm);
  hipFree(_mesh->faces);

  _mesh->vert  = nullptr;
  _mesh->norm  = nullptr;
  _mesh->faces = nullptr;

  _mesh->numVert  = 0;
  _mesh->numNorm  = 0;
  _mesh->numFaces = 0;

  return lRes == hipSuccess;
}

// __global__ void findTopKthElementDevice(DeviceTensor<float, 1> _data, uint32_t k, float *_out) {
//   float topK = warpFindTopKthElement(_data, k).k;
//
//   if (threadIdx.x == 0) {
//     *_out = topK;
//   }
// }


template <typename T>
struct results_t {
  float time;
  T     val;
};

template <typename T>
void setupForTiming(hipEvent_t &start, hipEvent_t &stop /*, T **d_vec, T* h_vec, uint size*/, results_t<T> **result) {
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //   hipMalloc(d_vec, size * sizeof(T));
  //   hipMemcpy(*d_vec, h_vec, size * sizeof(T), hipMemcpyHostToDevice);
  *result = (results_t<T> *)malloc(sizeof(results_t<T>));
}

template <typename T>
void wrapupForTiming(hipEvent_t &start, hipEvent_t &stop /*, T* d_vec*/, results_t<T> *result, float time, T value) {
  //   hipFree(d_vec);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  result->val  = value;
  result->time = time;
  //   hipDeviceSynchronize();
}

extern "C" float topKThElement(float *_data, uint32_t size, uint32_t k) {
  //   float *dResult = nullptr;
  //   float lResult = 0.0f;
  //
  //   hipMalloc(&dResult, 1 * sizeof(float));
  //
  //   int dataSizes[] = { (int) _num };
  //
  //   findTopKthElementDevice<<<1, 32>>>(DeviceTensor<float, 1>(_data, dataSizes), _k, dResult);
  //
  //   hipMemcpy(&lResult, dResult, 1 * sizeof(float), hipMemcpyDeviceToHost);
  //   hipFree(dResult);
  //   return lResult;

  hipEvent_t       start, stop;
  float             time;
  results_t<float> *result;
  float             retFromSelect;
  float *           deviceVec = _data;
  hipDeviceProp_t    dp;
  hipGetDeviceProperties(&dp, 0);


  setupForTiming(start, stop /*, &deviceVec, hostVec, size*/, &result);

  hipEventRecord(start, 0);

  retFromSelect = bucketSelectWrapper(deviceVec, size, k, dp.multiProcessorCount, dp.maxThreadsPerBlock);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);


  wrapupForTiming(start, stop /*, deviceVec*/, result, time, retFromSelect);
  //   return result;

  return retFromSelect;
}



template <typename T>
void setupForTimingH(hipEvent_t &start, hipEvent_t &stop, T **d_vec, T *h_vec, uint size, results_t<T> **result) {
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMalloc(d_vec, size * sizeof(T));
  hipMemcpy(*d_vec, h_vec, size * sizeof(T), hipMemcpyHostToDevice);
  *result = (results_t<T> *)malloc(sizeof(results_t<T>));
}

template <typename T>
void wrapupForTimingH(hipEvent_t &start, hipEvent_t &stop, T *d_vec, results_t<T> *result, float time, T value) {
  hipFree(d_vec);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  result->val  = value;
  result->time = time;
  //   hipDeviceSynchronize();
}

extern "C" float topKThElementHost(float *_data, uint32_t _num, uint32_t _k) {
  hipEvent_t       start, stop;
  float             time;
  results_t<float> *result;
  float             retFromSelect;
  float *           deviceVec;
  hipDeviceProp_t    dp;
  hipGetDeviceProperties(&dp, 0);


  setupForTimingH(start, stop, &deviceVec, _data, _num, &result);

  hipEventRecord(start, 0);

  retFromSelect = BucketSelect::bucketSelectWrapper(deviceVec, _num, _k, dp.multiProcessorCount, dp.maxThreadsPerBlock);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);


  wrapupForTimingH(start, stop, deviceVec, result, time, retFromSelect);
  return retFromSelect;
}

extern "C" bool runMalloc(void **_ptr, size_t _size) {
  hipError_t lRes;
  CUDA_RUN(hipMalloc(_ptr, _size));

  return true;
error:
  return false;
}

extern "C" bool runMemcpy(void *_dest, void *_src, size_t _size, MemcpyKind _kind) {
  hipError_t lRes;
  CUDA_RUN(hipMemcpy(_dest, _src, _size, static_cast<hipMemcpyKind>(_kind)));

  return true;
error:
  return false;
}

extern "C" void runFree(void *_ptr) { hipFree(_ptr); }

extern "C" __global__ void kTransformVecs(vec3 *_src, vec3 *_dest, uint32_t _size, mat4 _mat) {
  for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < _size; i += blockDim.x * gridDim.x) {
    _dest[i] = _mat * vec4(_src[i], 1.0f);
  }
}

extern "C" void transformVecs(vec3 *_src, vec3 *_dest, uint32_t _size, mat4 _mat) {
  kTransformVecs<<<(_size + 256 - 1) / 256, 256>>>(_src, _dest, _size, _mat);
}
