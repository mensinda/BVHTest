#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2018 Daniel Mensinger
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudaFN.hpp"
#include <iostream>

using namespace glm;
using namespace std;
using namespace BVHTest;
using namespace BVHTest::base;
using namespace BVHTest::cuda;

#define CUDA_RUN(call)                                                                                                 \
  lRes = call;                                                                                                         \
  if (lRes != hipSuccess) { goto error; }

extern "C" bool copyBVHToGPU(BVH *_bvh, BVH **_ptr) {
  if (!_bvh) { return false; }

  BVHNode *   lData = nullptr;
  size_t      lSize = _bvh->size() * sizeof(BVHNode);
  BVH         lTempBVH;
  hipError_t lRes;

  CUDA_RUN(hipMalloc(&lData, lSize));
  CUDA_RUN(hipMemcpy(lData, _bvh->data(), lSize, hipMemcpyHostToDevice));

  lTempBVH.setNewRoot(_bvh->root());
  lTempBVH.setMaxLevel(_bvh->maxLevel());
  lTempBVH.setMemory(lData, _bvh->size(), _bvh->size());

  CUDA_RUN(hipMalloc(_ptr, sizeof(BVH)));
  CUDA_RUN(hipMemcpy(*_ptr, &lTempBVH, sizeof(BVH), hipMemcpyHostToDevice));

  lTempBVH.setMemory(nullptr, 0, 0); // Avoid destructor segfault

  return true;

error:
  if (lData) { hipFree(lData); }
  if (*_ptr) { hipFree(*_ptr); }

  lTempBVH.setMemory(nullptr, 0, 0); // Avoid destructor segfault

  *_ptr = nullptr;
  return false;
}

extern "C" bool copyMeshToGPU(Mesh *_mesh, MeshRaw *_meshOut) {
  if (!_mesh) { return false; }

  hipError_t lRes;
  uint32_t    lVertSize  = _mesh->vert.size() * sizeof(vec3);
  uint32_t    lNromSize  = _mesh->norm.size() * sizeof(vec3);
  uint32_t    lFacesSize = _mesh->faces.size() * sizeof(Triangle);

  CUDA_RUN(hipMalloc(&_meshOut->vert, lVertSize));
  CUDA_RUN(hipMalloc(&_meshOut->norm, lNromSize));
  CUDA_RUN(hipMalloc(&_meshOut->faces, lFacesSize));

  CUDA_RUN(hipMemcpy(_meshOut->vert, _mesh->vert.data(), lVertSize, hipMemcpyHostToDevice));
  CUDA_RUN(hipMemcpy(_meshOut->norm, _mesh->norm.data(), lNromSize, hipMemcpyHostToDevice));
  CUDA_RUN(hipMemcpy(_meshOut->faces, _mesh->faces.data(), lFacesSize, hipMemcpyHostToDevice));

  _meshOut->numVert  = _mesh->vert.size();
  _meshOut->numNorm  = _mesh->norm.size();
  _meshOut->numFaces = _mesh->faces.size();

  return true;

error:
  if (_meshOut->vert) { hipFree(_meshOut->vert); }
  if (_meshOut->norm) { hipFree(_meshOut->norm); }
  if (_meshOut->faces) { hipFree(_meshOut->faces); }

  _meshOut->vert  = nullptr;
  _meshOut->norm  = nullptr;
  _meshOut->faces = nullptr;
  return false;
}



extern "C" bool copyBVHToHost(base::BVH **_bvh, base::BVH *_ptr) {
  if (!_bvh || !*_bvh || !_ptr) { return false; }

  hipError_t lRes;
  BVH         lTempBVH;
  uint32_t    lSize   = 0;
  BVHNode *   lSource = nullptr;

  CUDA_RUN(hipMemcpy(&lTempBVH, *_bvh, sizeof(BVH), hipMemcpyDeviceToHost));

  _ptr->setNewRoot(lTempBVH.root());
  _ptr->setMaxLevel(lTempBVH.maxLevel());
  _ptr->resize(lTempBVH.size());

  lSize   = lTempBVH.size() * sizeof(BVHNode);
  lSource = lTempBVH.data();

  CUDA_RUN(hipMemcpy(_ptr->data(), lSource, lSize, hipMemcpyDeviceToHost));

error:
  hipFree(_ptr);
  hipFree(lSource);

  lTempBVH.setMemory(nullptr, 0, 0); // Avoid destructor segfault
  *_bvh = nullptr;

  return lRes == hipSuccess;
}


extern "C" bool copyMeshToHost(base::MeshRaw *_mesh, base::Mesh *_meshOut) {
  if (!_mesh || !_mesh->vert || !_mesh->norm || !_mesh->faces || !_meshOut) { return false; }

  hipError_t lRes;

  _meshOut->vert.resize(_mesh->numVert);
  _meshOut->norm.resize(_mesh->numNorm);
  _meshOut->faces.resize(_mesh->numFaces);

  uint32_t lVertSize  = _mesh->numVert * sizeof(vec3);
  uint32_t lNromSize  = _mesh->numNorm * sizeof(vec3);
  uint32_t lFacesSize = _mesh->numFaces * sizeof(Triangle);

  CUDA_RUN(hipMemcpy(_meshOut->vert.data(), _mesh->vert, lVertSize, hipMemcpyDeviceToHost));
  CUDA_RUN(hipMemcpy(_meshOut->norm.data(), _mesh->norm, lNromSize, hipMemcpyDeviceToHost));
  CUDA_RUN(hipMemcpy(_meshOut->faces.data(), _mesh->faces, lFacesSize, hipMemcpyDeviceToHost));

error:
  hipFree(_mesh->vert);
  hipFree(_mesh->norm);
  hipFree(_mesh->faces);

  _mesh->vert  = nullptr;
  _mesh->norm  = nullptr;
  _mesh->faces = nullptr;

  _mesh->numVert  = 0;
  _mesh->numNorm  = 0;
  _mesh->numFaces = 0;

  return lRes == hipSuccess;
}
