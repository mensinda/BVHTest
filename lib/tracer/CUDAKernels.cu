#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2018 Daniel Mensinger
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#define GLM_ENABLE_EXPERIMENTAL

#include <base/Ray.hpp>
#include <glm/gtx/intersect.hpp>
#include <glm/gtx/normal.hpp>
#include <glm/mat4x4.hpp>
#include "CUDAKernels.hpp"
#include <glm/gtc/matrix_transform.hpp>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#define CUDA_RUN(call)                                                                                                 \
  lRes = call;                                                                                                         \
  if (lRes != hipSuccess) {                                                                                           \
    cout << "CUDA ERROR (" << __FILE__ << ":" << __LINE__ << "): " << hipGetErrorString(lRes) << endl;                \
    goto error;                                                                                                        \
  }

using namespace std;
using namespace glm;
using namespace BVHTest;
using namespace BVHTest::base;

extern "C" __global__ void kGenerateRays(
    Ray *_rays, uint32_t _w, uint32_t _h, mat4 _mat, vec3 _pos, float _ratio, float _scale) {
  uint32_t iX = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t iY = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t sX = blockDim.x * gridDim.x;
  uint32_t sY = blockDim.y * gridDim.y;

  for (uint32_t y = iY; y < _h; y += sY) {
    for (uint32_t x = iX; x < _w; x += sX) {
      float lPixX      = (2 * ((x + 0.5) / _w) - 1) * _scale * _ratio;
      float lPixY      = (1 - 2 * ((y + 0.5) / _h)) * _scale;
      vec3  lDirection = _mat * vec4(lPixX, lPixY, -1, 0.0f);

      _rays[y * _w + x].set(_pos, normalize(lDirection), x, y);
    }
  }
}

struct DataShared {
  Triangle closest;
  float    nearest;

  //   vec3      origin;
  //   vec3      direction;
  //   vec3      invDir;
  //   Ray::Sign sign;
};

extern "C" __device__ __forceinline__ bool intersectRayAABB(
    AABB &_aabb, Ray const &_r, float t0, float t1, float &tmin, float &tmax) {
  glm::vec3 const &lOrigin = _r.getOrigin();
  glm::vec3 const &lInvDir = _r.getInverseDirection();
  Ray::Sign const &lSign   = _r.getSign();

  float tymin, tymax, tzmin, tzmax;

  tmin  = (_aabb.minMax[lSign.x].x - lOrigin.x) * lInvDir.x;
  tmax  = (_aabb.minMax[1 - lSign.x].x - lOrigin.x) * lInvDir.x;
  tymin = (_aabb.minMax[lSign.y].y - lOrigin.y) * lInvDir.y;
  tymax = (_aabb.minMax[1 - lSign.y].y - lOrigin.y) * lInvDir.y;
  tzmin = (_aabb.minMax[lSign.z].z - lOrigin.z) * lInvDir.z;
  tzmax = (_aabb.minMax[1 - lSign.z].z - lOrigin.z) * lInvDir.z;
  if (tymin > tmin) { tmin = tymin; }
  if (tzmin > tmin) { tmin = tzmin; }
  if (tymax < tmin) { tmax = tymax; }
  if (tzmax < tmin) { tmax = tzmax; }

  return ((tmin < tmax) && (tmin < t1) && (tmax > t0));
}

extern "C" __global__ void kTraceRay(Ray *    _rays,
                                     uint8_t *_img,
                                     BVHNode *_nodes,
                                     uint32_t _rootNode,
                                     MeshRaw  _mesh,
                                     vec3     _light,
                                     uint32_t _w,
                                     uint32_t _h) {
  uint32_t iX = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t iY = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t sX = blockDim.x * gridDim.x;
  uint32_t sY = blockDim.y * gridDim.y;

  uint32_t lID = threadIdx.y * 8 + threadIdx.x;

  for (uint32_t y = iY; y < _h; y += sY) {
    for (uint32_t x = iX; x < _w; x += sX) {
      Ray       lRay = _rays[y * _w + x];
      CUDAPixel lRes = {121, 167, 229, 0};

      __shared__ DataShared lEtcData[64];

      //       {
      //         Ray lRay                = _rays[y * _w + x];
      //         lEtcData[lID].origin    = lRay.getOrigin();
      //         lEtcData[lID].direction = lRay.getOrigin();
      //         lEtcData[lID].invDir    = lRay.getInverseDirection();
      //         lEtcData[lID].sign      = lRay.getSign();
      //       }

      /*
       * Algorithm from:
       *
       * Attila T. Áfra and László Szirmay-Kalos. “Stackless Multi-BVH Traversal for CPU,
       * MIC and GPU Ray Tracing”. In: Computer Graphics Forum 33.1 (2014), pp. 129–140.
       * doi: 10.1111/cgf.12259. eprint: https://onlinelibrary.wiley.com/doi/pdf/
       * 10.1111/cgf.12259. url: https://onlinelibrary.wiley.com/doi/abs/10.1111/
       * cgf.12259.
       */

      uint64_t lBitStack_lo = 0;
      uint64_t lBitStack_hi = 0;
      uint32_t lNode        = _rootNode;

      lEtcData[lID].nearest = HUGE_VALF;
      dvec2 lBarycentricTemp;

      float  lMinLeft;
      float  lMinRight;
      float  lTemp;
      double lDistance;

      while (true) {
        if (!_nodes[lNode].isLeaf()) {
          lRes.intCount++;
          uint32_t lLeft  = _nodes[lNode].left;
          uint32_t lRight = _nodes[lNode].right;
          bool     lLeftHit =
              intersectRayAABB(_nodes[lLeft].bbox, lRay, 0.01f, lEtcData[lID].nearest + 0.01f, lMinLeft, lTemp);
          bool lRightHit =
              intersectRayAABB(_nodes[lLeft].bbox, lRay, 0.01f, lEtcData[lID].nearest + 0.01f, lMinRight, lTemp);

          if (lLeftHit || lRightHit) {
            lBitStack_hi = (lBitStack_hi << 1) | (lBitStack_lo >> 63);
            lBitStack_lo <<= 1;

            if (lLeftHit && lRightHit) {
              lBitStack_lo |= 1;
              lNode = lMinLeft < lMinRight ? lLeft : lRight;
            } else {
              lNode = lLeftHit ? lLeft : lRight;
            }

            continue;
          }
        } else {
          for (uint32_t i = 0; i < _nodes[lNode].numFaces(); ++i) {
            Triangle lTri = _mesh.faces[_nodes[lNode].beginFaces() + i];

            bool lHit = intersectRayTriangle<double>(static_cast<dvec3 const &>(lRay.getOrigin()),
                                                     static_cast<dvec3 const &>(lRay.getDirection()),
                                                     static_cast<dvec3 const &>(_mesh.vert[lTri.v1]),
                                                     static_cast<dvec3 const &>(_mesh.vert[lTri.v2]),
                                                     static_cast<dvec3 const &>(_mesh.vert[lTri.v3]),
                                                     lBarycentricTemp,
                                                     lDistance);

            if (lHit && lDistance < lEtcData[lID].nearest) {
              lEtcData[lID].nearest = lDistance;
              lEtcData[lID].closest = lTri;
            }
          }
        }

        // Backtrac
        while ((lBitStack_lo & 1) == 0) {
          if (lBitStack_lo == 0 && lBitStack_hi == 0) { goto LABEL_END; } // I know, I know...
          lNode        = _nodes[lNode].parent;
          lBitStack_lo = (lBitStack_lo >> 1) | (lBitStack_hi << 63);
          lBitStack_hi >>= 1;
        }

        lNode = _nodes[lNode].isRightChild() ? _nodes[_nodes[lNode].parent].left : _nodes[_nodes[lNode].parent].right;
        lBitStack_lo ^= 1;
      }

    LABEL_END:

      if (lEtcData[lID].nearest < HUGE_VALF) {
        Triangle lClosest  = lEtcData[lID].closest;
        vec3     lNorm     = triangleNormal(_mesh.vert[lClosest.v1], _mesh.vert[lClosest.v2], _mesh.vert[lClosest.v3]);
        vec3     lHitPos   = lRay.getOrigin() + lEtcData[lID].nearest * lRay.getDirection();
        vec3     lLightDir = normalize(_light - lHitPos);
        float    lDiffuse  = 1.0f + dot(lNorm, lLightDir);
        lDiffuse           = lDiffuse > 0.0f ? lDiffuse : 0.0f;

        lRes.r = lRes.g = lRes.b = static_cast<uint8_t>(lDiffuse * 127.0f);
      }

      reinterpret_cast<CUDAPixel *>(_img)[y * _w + x] = lRes;
    }
  }
}

enum TRAV { TRAV_NONE = 0, TRAV_LEFT = 1, TRAV_RIGHT = 2, TRAV_BOTH = 3 };

extern "C" __device__ __forceinline__ void dReduce64Resolve(int32_t *_res, uint32_t _id) {
  __syncthreads();
  if (_id < 32) {
    _res[_id] += _res[_id + 32];
    _res[_id] += _res[_id + 16];
    _res[_id] += _res[_id + 8];
    _res[_id] += _res[_id + 4];
    _res[_id] += _res[_id + 2];
    _res[_id] += _res[_id + 1];
  }
  __syncthreads();
}

extern "C" __global__ void kTraceRayBundle(Ray *    _rays,
                                           uint8_t *_img,
                                           BVHNode *_nodes,
                                           uint32_t _rootNode,
                                           MeshRaw  _mesh,
                                           vec3     _light,
                                           uint32_t _w,
                                           uint32_t _h) {
  uint32_t iX = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t iY = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t sX = blockDim.x * gridDim.x;
  uint32_t sY = blockDim.y * gridDim.y;

  uint32_t lID = threadIdx.y * 8 + threadIdx.x;

  for (uint32_t y = iY; y < _h; y += sY) {
    for (uint32_t x = iX; x < _w; x += sX) {
      Ray       lRay = _rays[y * _w + x];
      CUDAPixel lRes = {121, 167, 229, 0};

      __shared__ BVHNode lNodes[3]; // 0: left // 1: right // 2: current
      __shared__ int32_t lResolve[64];
      __shared__ bool    lTravNext[4]; // 0: none // 1: left // 2: right // 3: both
      __shared__ DataShared lEtcData[64];

      uint64_t lBitStack_lo = 0;
      uint64_t lBitStack_hi = 0;
      uint32_t lNode        = _rootNode;

      lEtcData[lID].nearest = HUGE_VALF;

      while (true) {
        if (lID == 0) { lNodes[2] = _nodes[lNode]; }
        __syncthreads();

        uint32_t lChildren[2]; // 0: left // 1: right
        lChildren[0] = lNodes[2].left;
        lChildren[1] = lNodes[2].right;

        if (!lNodes[2].isLeaf()) {
          lRes.intCount++;

          if (lID < 2) { lNodes[lID] = _nodes[lChildren[lID]]; } // Load children into shared memory
          __syncthreads();

          float lMinLeft;
          float lMinRight;
          float lTemp;

          uint32_t lLeftHit =
              intersectRayAABB(lNodes[0].bbox, lRay, 0.01f, lEtcData[lID].nearest + 0.01f, lMinLeft, lTemp);
          uint32_t lRightHit =
              intersectRayAABB(lNodes[1].bbox, lRay, 0.01f, lEtcData[lID].nearest + 0.01f, lMinRight, lTemp);

          if (lID < 4) { lTravNext[lID] = false; } // Reset trav next
          __syncthreads();
          lTravNext[lRightHit * 2 + lLeftHit] = true;
          __syncthreads();

          // ========================
          // = Check wat to do next =
          // ========================

          if (lTravNext[TRAV_BOTH] || (lTravNext[TRAV_LEFT] && lTravNext[TRAV_RIGHT])) {
            // Both hit somehow
            lBitStack_hi = (lBitStack_hi << 1) | (lBitStack_lo >> 63);
            lBitStack_lo <<= 1;
            lBitStack_lo |= 1;

            // Set what this node wants
            lResolve[lID] = (lLeftHit && (lMinLeft < lMinRight || !lRightHit)) - // Left is prefered --> 1
                            (lRightHit && (lMinRight < lMinLeft || !lLeftHit));  // Right is prefered --> -1

            dReduce64Resolve(lResolve, lID);

            if (lResolve[0] > 0) { // Left is prefered
              lNode = lChildren[0];
            } else { // Right is prefered
              lNode = lChildren[1];
            }

            continue;
          } else if (lTravNext[TRAV_LEFT]) {
            lBitStack_hi = (lBitStack_hi << 1) | (lBitStack_lo >> 63);
            lBitStack_lo <<= 1;
            lNode = lChildren[0];
            continue;
          } else if (lTravNext[TRAV_RIGHT]) {
            lBitStack_hi = (lBitStack_hi << 1) | (lBitStack_lo >> 63);
            lBitStack_lo <<= 1;
            lNode = lChildren[1];
            continue;
          }
        } else {
          for (uint32_t i = 0; i < lChildren[1]; ++i) {
            Triangle lTri = _mesh.faces[lChildren[0] + i];

            dvec2  lBarycentricTemp;
            double lDistance;

            bool lHit = intersectRayTriangle<double>(static_cast<dvec3 const &>(lRay.getOrigin()),
                                                     static_cast<dvec3 const &>(lRay.getDirection()),
                                                     static_cast<dvec3 const &>(_mesh.vert[lTri.v1]),
                                                     static_cast<dvec3 const &>(_mesh.vert[lTri.v2]),
                                                     static_cast<dvec3 const &>(_mesh.vert[lTri.v3]),
                                                     lBarycentricTemp,
                                                     lDistance);

            if (lHit && lDistance < lEtcData[lID].nearest) {
              lEtcData[lID].nearest = lDistance;
              lEtcData[lID].closest = lTri;
            }
          }
        }

        // Backtrac
        while ((lBitStack_lo & 1) == 0) {
          if (lBitStack_lo == 0 && lBitStack_hi == 0) { goto LABEL_END; } // I know, I know...
          lNode        = lNodes[2].parent;
          lBitStack_lo = (lBitStack_lo >> 1) | (lBitStack_hi << 63);
          lBitStack_hi >>= 1;

          if (lID == 0) { lNodes[2] = _nodes[lNode]; }
          __syncthreads();
        }

        lNode = lNodes[2].isRightChild() ? _nodes[lNodes[2].parent].left : _nodes[lNodes[2].parent].right;
        lBitStack_lo ^= 1;
      }

    LABEL_END:

      if (lEtcData[lID].nearest < HUGE_VALF) {
        Triangle lClosest  = lEtcData[lID].closest;
        vec3     lNorm     = triangleNormal(_mesh.vert[lClosest.v1], _mesh.vert[lClosest.v2], _mesh.vert[lClosest.v3]);
        vec3     lHitPos   = lRay.getOrigin() + lEtcData[lID].nearest * lRay.getDirection();
        vec3     lLightDir = normalize(_light - lHitPos);
        float    lDiffuse  = 1.0f + dot(lNorm, lLightDir);
        lDiffuse           = lDiffuse > 0.0f ? lDiffuse : 0.0f;

        lRes.r = lRes.g = lRes.b = static_cast<uint8_t>(lDiffuse * 127.0f);
      }

      reinterpret_cast<CUDAPixel *>(_img)[y * _w + x] = lRes;
    }
  }
}


extern "C" void generateRays(Ray *_rays, uint32_t _w, uint32_t _h, vec3 _pos, vec3 _lookAt, vec3 _up, float _fov) {
  if (!_rays) { return; }

  mat4  lCamToWorld  = inverse(lookAtRH(_pos, _lookAt, _up));
  float lAspectRatio = static_cast<float>(_w) / static_cast<float>(_h);
  float lScale       = tan(radians(0.5 * _fov));

  dim3 lBlock(16, 16, 1);
  dim3 lGrid((_w + lBlock.x - 1) / lBlock.x, (_h + lBlock.y - 1) / lBlock.y);

  kGenerateRays<<<lGrid, lBlock>>>(_rays, _w, _h, lCamToWorld, _pos, lAspectRatio, lScale);
}

extern "C" void tracerImage(Ray *    _rays,
                            uint8_t *_img,
                            BVHNode *_nodes,
                            uint32_t _rootNode,
                            MeshRaw  _mesh,
                            vec3     _light,
                            uint32_t _w,
                            uint32_t _h,
                            bool     _bundle) {
  if (!_rays || !_img) { return; }
  dim3 lBlock(8, 8, 1);
  dim3 lGrid((_w + lBlock.x - 1) / lBlock.x, (_h + lBlock.y - 1) / lBlock.y);

  if (_bundle) {
    kTraceRayBundle<<<lGrid, lBlock>>>(_rays, _img, _nodes, _rootNode, _mesh, _light, _w, _h);
  } else {
    kTraceRay<<<lGrid, lBlock>>>(_rays, _img, _nodes, _rootNode, _mesh, _light, _w, _h);
  }
}

extern "C" bool copyToOGLImage(void **_resource, uint8_t *_img, uint32_t _w, uint32_t _h) {
  hipError_t            lRes;
  hipArray_t            lDevArray;
  hipGraphicsResource **lResource = reinterpret_cast<hipGraphicsResource **>(_resource);

  CUDA_RUN(hipPeekAtLastError());
  CUDA_RUN(hipGraphicsMapResources(1, lResource, 0));
  CUDA_RUN(hipGraphicsSubResourceGetMappedArray(&lDevArray, *lResource, 0, 0));

  CUDA_RUN(hipMemcpyToArray(lDevArray, 0, 0, _img, _w * _h * 4 * sizeof(uint8_t), hipMemcpyDeviceToDevice));

  CUDA_RUN(hipGraphicsUnmapResources(1, lResource, 0));

  return true;

error:
  return false;
}

extern "C" void copyImageToHost(CUDAPixel *_hostPixel, uint8_t *_cudaImg, uint32_t _w, uint32_t _h) {
  hipMemcpy(_hostPixel, _cudaImg, _w * _h * sizeof(CUDAPixel), hipMemcpyDeviceToHost);
}


extern "C" bool registerOGLImage(void **_resource, uint32_t _image) {
  hipError_t lRes;
  CUDA_RUN(hipGraphicsGLRegisterImage(reinterpret_cast<hipGraphicsResource **>(_resource),
                                       _image,
                                       GL_TEXTURE_2D,
                                       hipGraphicsRegisterFlagsWriteDiscard));
  return true;

error:
  return false;
}

extern "C" void unregisterOGLImage(void *_resource) {
  hipGraphicsUnregisterResource(reinterpret_cast<hipGraphicsResource *>(_resource));
}


extern "C" bool allocateRays(Ray **_rays, uint32_t _num) {
  hipError_t lRes;
  CUDA_RUN(hipMalloc(_rays, _num * sizeof(Ray)));
  return true;

error:
  return false;
}

extern "C" bool allocateImage(uint8_t **_img, uint32_t _w, uint32_t _h) {
  hipError_t lRes;
  CUDA_RUN(hipMalloc(_img, _w * _h * sizeof(CUDAPixel)));
  return true;

error:
  return false;
}

extern "C" void freeRays(Ray **_rays) {
  if (*_rays) {
    hipFree(*_rays);
    *_rays = NULL;
  }
}

extern "C" void freeImage(uint8_t *_img) {
  if (_img) { hipFree(_img); }
}

extern "C" void tracerDoCudaSync() { hipDeviceSynchronize(); }
extern "C" void initCUDA_GL() {
  hipError_t lRes;
  CUDA_RUN(cudaGLSetGLDevice(0));
error:
  return;
}
